#include "gpu_core.h"
#include <hip/hip_runtime.h>
#include "kernels.cuh"

#if defined(DEBUG) || defined(_DEBUG)
    #include <stdio.h>
    #include <assert.h>
#endif
//-----------------------------------------------------------------------------
// Forward declarations
int getNumSMs();
inline hipError_t checkCuda(hipError_t result);
//-----------------------------------------------------------------------------
const bool gpu_available()
{
    int deviceCount;
    hipError_t errorId = hipGetDeviceCount(&deviceCount);

    return errorId == hipSuccess
        && deviceCount > 0;
}
//-----------------------------------------------------------------------------
const int gpu_vector_add(double* a, double* b, double* c, const int n)
{
    double* dA;
    double* dB;
    double* dC;

    const int size = sizeof(double) * n;

    try
    {
        checkCuda(hipMalloc(&dA, size));
        checkCuda(hipMalloc(&dB, size));
        checkCuda(hipMalloc(&dC, size));

        checkCuda(hipMemcpy(dA, a, size, hipMemcpyHostToDevice));
        checkCuda(hipMemcpy(dB, b, size, hipMemcpyHostToDevice));
        checkCuda(hipMemcpy(dC, c, size, hipMemcpyHostToDevice));

        const int blockSize = 256;
        int numBlocks = (n + blockSize - 1) / blockSize;

#if defined(DEBUG) || defined(_DEBUG)
        printf("blockSize: %d\nnumBlocks: %d\n", blockSize, numBlocks);
#endif

        addKernel<<< numBlocks, blockSize >>> (dA, dB, dC, n);

        checkCuda(hipMemcpy(c, dC, size, hipMemcpyDeviceToHost));

        checkCuda(hipDeviceSynchronize());

        checkCuda(hipFree(dA));
        checkCuda(hipFree(dB));
        checkCuda(hipFree(dC));
    }
    catch (const int e)
    {
        return e;
    }

    return 0;
}
//-----------------------------------------------------------------------------
const char* gpu_get_error_string(const int errorCode)
{
    return hipGetErrorString(static_cast<hipError_t>(errorCode));
}
//-----------------------------------------------------------------------------
int getNumSMs()
{
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    return numSMs;
}
//-----------------------------------------------------------------------------
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));

        assert(result == hipSuccess);
    }
#else
    if (result != hipSuccess)
    {
        throw static_cast<int>(result);
    }
#endif

    return result;
}
